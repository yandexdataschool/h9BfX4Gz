#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cutlass/gemm/device/gemm.h>


torch::Tensor int4MatmulCUDA(const torch::Tensor &A, const torch::Tensor &B) {
  torch::checkAllSameGPU("int4Matmul", {{A, "A", 0}, {B, "B", 1}});
  auto M = A.size(0);
  auto N = B.size(0);
  auto K = A.size(1) * 2;  // = B.size(1) * 2 . 4bit packing is on the columns
  auto C = torch::empty({M, N}, torch::dtype(torch::kInt32).device(A.device()));

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::int4b_t,                // ElementA
      cutlass::layout::RowMajor,       // LayoutA
      cutlass::int4b_t,                // ElementB
      cutlass::layout::ColumnMajor,    // LayoutB
      int32_t,                         // ElementOutput
      cutlass::layout::RowMajor,       // LayoutOutput
      int32_t,                         // ElementAccumulator
      cutlass::arch::OpClassTensorOp,  // tag indicating Tensor Cores
      cutlass::arch::Sm75  // tag indicating target GPU compute architecture
      >;

  Gemm gemmOp;

  using GemmCoord = cutlass::gemm::GemmCoord;

  typename Gemm::Arguments arguments{
      {static_cast<GemmCoord::Index>(M), static_cast<GemmCoord::Index>(N),
       static_cast<GemmCoord::Index>(K)},
      {(cutlass::int4b_t *)A.data_ptr<uint8_t>(), K},
      {(cutlass::int4b_t *)B.data_ptr<uint8_t>(), K},
      {C.data_ptr<int32_t>(), N},
      {C.data_ptr<int32_t>(), N},
      {1, 0}};

  auto status = gemmOp(arguments);

  TORCH_CHECK(status == cutlass::Status::kSuccess,
              cutlassGetStatusString(status))

  return C;
}

torch::Tensor int8MatmulCUDA(const torch::Tensor &A, const torch::Tensor &B) {
  torch::checkAllSameGPU("int8Matmul", {{A, "A", 0}, {B, "B", 1}});
  auto M = A.size(0);
  auto N = B.size(0);
  auto K = A.size(1);  // = B.size(1)
  auto C = torch::empty({M, N}, torch::dtype(torch::kInt32).device(A.device()));

  using Gemm = cutlass::gemm::device::Gemm<
      int8_t,                          // ElementA
      cutlass::layout::RowMajor,       // LayoutA
      int8_t,                          // ElementB
      cutlass::layout::ColumnMajor,    // LayoutB
      int32_t,                         // ElementOutput
      cutlass::layout::RowMajor,       // LayoutOutput
      int32_t,                         // ElementAccumulator
      cutlass::arch::OpClassTensorOp,  // tag indicating Tensor Cores
      cutlass::arch::Sm75  // tag indicating target GPU compute architecture
      >;

  Gemm gemmOp;

  using GemmCoord = cutlass::gemm::GemmCoord;

  typename Gemm::Arguments arguments{
      {static_cast<GemmCoord::Index>(M), static_cast<GemmCoord::Index>(N),
       static_cast<GemmCoord::Index>(K)},
      {A.data_ptr<int8_t>(), K},
      {B.data_ptr<int8_t>(), K},
      {C.data_ptr<int32_t>(), N},
      {C.data_ptr<int32_t>(), N},
      {1, 0}};

  auto status = gemmOp(arguments);

  TORCH_CHECK(status == cutlass::Status::kSuccess,
              cutlassGetStatusString(status))

  return C;
}
